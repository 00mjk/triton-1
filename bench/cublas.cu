#include <ctime>
#include <cstdio>
#include <common_functions.h>
#include <hipblas.h>
#include <vector>
#include <algorithm>
#include <iostream>

#include "common.hpp"

template<class NumericT>
void run()
{
    #define FILL_TIMINGS(OP, timings) \
    {\
      float elapsedTime = 0; \
      float total_time = 0; \
      OP;\
      while(total_time < 1e-1) \
      { \
        hipEvent_t start, stop; \
        hipEventCreate(&start); \
        hipEventRecord(start,0); \
        OP; \
        hipDeviceSynchronize(); \
        hipEventCreate(&stop); \
        hipEventRecord(stop,0); \
        hipEventSynchronize(stop); \
        hipEventElapsedTime(&elapsedTime, start,stop); \
        timings.push_back(elapsedTime/1e3); \
        total_time += elapsedTime/1e3; \
      }\
    }

    //AXPY
    std::cout << "#vector-axpy" << std::endl;
    std::cout << "#N Perf" << std::endl;
    for(std::vector<int>::const_iterator it = BLAS1_N.begin() ; it != BLAS1_N.end() ; ++it)
    {
      int N = *it;
      NumericT *x, *y;
      hipMalloc((void**) &x, N * sizeof(NumericT));
      hipMalloc((void**) &y, N * sizeof(NumericT));
      //Bench
      std::vector<float> timings;
      FILL_TIMINGS(hipblasSaxpy(N, 2, x, 1, y, 1), timings);
      std::cout << N << " " << 3*N*sizeof(NumericT)*1e-9/median(timings) << std::endl;
      //Free
      hipFree(x);
      hipFree(y);
    }
    std::cout << std::endl;
    std::cout << std::endl;

    std::cout << "#dot" << std::endl;
    std::cout << "#N Perf" << std::endl;
    for(std::vector<int>::const_iterator it = BLAS1_N.begin() ; it != BLAS1_N.end() ; ++it)
    {
      int N = *it;
      NumericT *x, *y;
      hipMalloc((void**) &x, N * sizeof(NumericT));
      hipMalloc((void**) &y, N * sizeof(NumericT));
      //Bench
      std::vector<float> timings;
      FILL_TIMINGS(hipblasSdot(N, x, 1, y, 1), timings);
      std::cout << N << " " << 2*N*sizeof(NumericT)*1e-9/median(timings) << std::endl;
      //Free
      hipFree(x);
      hipFree(y);
    }
    std::cout << std::endl;
    std::cout << std::endl;


    std::cout << "#GEMV" << std::endl;
    std::cout << "#N Perf" << std::endl;
    for(std::vector<int>::const_iterator Mit = BLAS2_M.begin() ; Mit != BLAS2_M.end() ; ++Mit)
    {
        for(std::vector<int>::const_iterator it = BLAS2_N.begin() ; it != BLAS2_N.end() ; ++it)
        {
          int M = *Mit;
          int N = *it;
          NumericT *x, *y, *A;
          hipMalloc((void**) &A, M * N * sizeof(NumericT));
          hipMalloc((void**) &x, M * sizeof(NumericT));
          hipMalloc((void**) &y, N * sizeof(NumericT));
          //Bench
          std::vector<float> timings;
          FILL_TIMINGS(hipblasSgemv('N', M, N, 1.0, A, M, x, 1, 1.0, y, 1), timings);
          std::cout << N << " " << (M + N + M*N)*sizeof(NumericT)*1e-9/median(timings) << std::endl;
          //Free
          hipFree(A);
          hipFree(x);
          hipFree(y);
        }
        std::cout << std::endl;
        std::cout << std::endl;
    }

    std::cout << "#GEMM" << std::endl;
    std::cout << "#N Perf" << std::endl;
    for(std::vector<int>::const_iterator it = BLAS3_N.begin() ; it != BLAS3_N.end() ; ++it)
    {
      int M = *it;
      int N = *it;
      int K = *it;
      NumericT *C, *A, *B;
      hipMalloc((void**) &A, M * K * sizeof(NumericT));
      hipMalloc((void**) &B, K * N * sizeof(NumericT));
      hipMalloc((void**) &C, M * N * sizeof(NumericT));
      //Bench
      std::vector<float> timings;
      FILL_TIMINGS(hipblasSgemm('N','T',M,N,K,1.0,A,M,B,K,1.0,C,M), timings);
      std::cout << N << " " << 2.0*M*N*K*1e-9/median(timings) << std::endl;
      //Free
      hipFree(A);
      hipFree(B);
      hipFree(C);
    }
    std::cout << std::endl;
    std::cout << std::endl;
}

int main(int argc, char** argv)
{
  hipSetDevice(0);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  std::cout << "Device: " << prop.name << std::endl;
  run<float>();
}
